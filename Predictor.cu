#include "hip/hip_runtime.h"

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <iomanip>
#include <iostream>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>

#define CUDA_CALL(f) { \
  hipError_t err = (f); \
  if (err != hipSuccess) { \
    std::cout \
        << "    Error occurred: " << err << std::endl; \
    std::exit(1); \
  } \
}

#define CUDNN_CALL(f) { \
  hipdnnStatus_t err = (f); \
  if (err != HIPDNN_STATUS_SUCCESS) { \
    std::cout \
        << "    Error occurred: " << err << std::endl; \
    std::exit(1); \
  } \
}

__global__ void dev_const(float *px, float k) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  px[tid] = k;
}

__global__ void dev_inverse(float *px, float *py) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  px[tid] = 1/py[tid];
}

__global__ void dev_iota_bias(float *px, float k) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  px[tid] = tid*k;
}

__global__ void dev_iota(float *px) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  px[tid] = tid;
}

__global__ void doubleTofloat(const double* px, float *py) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  py[tid] = (float)px[tid];
}

__global__ void floatTodouble(const float* px, double *py) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  py[tid] = (double)px[tid];
}

__global__ void error(const float* px, const float* py, float *pz) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  pz[tid] = px[tid]-py[tid];
  }
  
  __global__ void addValues(float* px, float* py) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  px[tid] = px[tid]+py[tid];
  }

void print(const float *data, int n, int c, int h, int w) {
  std::vector<float> buffer(1 << 20);
  CUDA_CALL(hipMemcpy(
        buffer.data(), data,
        n * c * h * w * sizeof(float),
        hipMemcpyDeviceToHost));
  int a = 0;
  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < c; ++j) {
      std::cout << "n=" << i << ", c=" << j << ":" << std::endl;
      for (int k = 0; k < h; ++k) {
        for (int l = 0; l < w; ++l) {
          std::cout << std::setw(15) << std::right << buffer[a];
          ++a;
        }
        std::cout << std::endl;
      }
    }
  }
  std::cout << std::endl;
}

void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[]) {	
	char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";				 
	//CUBLAS and CUDNN handles
	hipdnnHandle_t cudnn;
	CUDNN_CALL(hipdnnCreate(&cudnn));
	hipblasHandle_t cublasHandle;
	hipblasCreate(&cublasHandle);

	//INPUT AND OUTPUT DATA
	//input
	mxGPUArray const* A;
	double const* d_A;
	double const* params;
	/* Throw an error if the input is not a GPU array. */
	if (!(mxIsGPUArray(prhs[0])) || (mxIsGPUArray(prhs[1]))) {
		mexErrMsgIdAndTxt(errId, "First argument must be GPUArray, second must not be.");
	}
	if(nrhs > 6){
		mexErrMsgIdAndTxt(errId, "Too many input arguments!");
	}
	
	A = mxGPUCreateFromMxArray(prhs[0]);
	//underlying pointer to input data
	d_A = (double const*)(mxGPUGetDataReadOnly(A));	
	//read and allocate input data
	int N = (int)(mxGPUGetNumberOfElements(A));;
	float* in_data;
	hipMallocManaged(&in_data,N*sizeof(float));
	//read parameters
	params = (double const*)(mxGetData(prhs[1]));
	int verbose = params[5];
	//read weights
	mxGPUArray const* weights_in[4];
	double const *d_weights_in[4];
	
	weights_in[0] = mxGPUCreateFromMxArray(prhs[2]);
	weights_in[1] = mxGPUCreateFromMxArray(prhs[3]);
	weights_in[2] = mxGPUCreateFromMxArray(prhs[4]);
	weights_in[3] = mxGPUCreateFromMxArray(prhs[5]);
	//pFC
	d_weights_in[0] = (double const*)(mxGPUGetDataReadOnly(weights_in[0]));
	//pFC_bias
	d_weights_in[1] = (double const*)(mxGPUGetDataReadOnly(weights_in[1]));
	//pOUT
	d_weights_in[2] = (double const*)(mxGPUGetDataReadOnly(weights_in[2]));
	//pOUT_bias
	d_weights_in[3] = (double const*)(mxGPUGetDataReadOnly(weights_in[3]));
	
	//output
	mxGPUArray* OUT;
	double* d_OUT;
	/* Create a GPUArray to hold the result and get its underlying pointer. */
	mwSize returnSize[] = {mxGPUGetDimensions(A)[0],mxGPUGetDimensions(A)[1]};
    OUT = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(A),
                            returnSize,
                            mxGPUGetClassID(A),
                            mxGPUGetComplexity(A),
                            MX_GPU_DO_NOT_INITIALIZE);
    d_OUT = (double *)(mxGPUGetData(OUT));
	
	// in_data
	const int in_n = params[0];
	const int in_h = params[1];
	const int in_w = params[2];
	if(verbose == 1){
		std::cout << "in_n: " << in_n << std::endl;
		std::cout << "in_h: " << in_h << std::endl;
		std::cout << "in_w: " << in_w << std::endl;
		std::cout << std::endl;
	}
	

	hipdnnTensorDescriptor_t in_desc;
	CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
	CUDNN_CALL(hipdnnSetTensor4dDescriptor(
		in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
		in_n, 1, in_h, in_w));
	

	  
	  
	//FC
	const int FC_h = params[3];
	const int FC_w = params[4];
	if(verbose == 1){
		std::cout << "FC_h: " << FC_h << std::endl;
		std::cout << "FC_w: " << FC_w << std::endl;
		std::cout << std::endl;
	}
	hipdnnTensorDescriptor_t FCTensorDesc;
	CUDNN_CALL(hipdnnCreateTensorDescriptor(&FCTensorDesc));
	CUDNN_CALL(hipdnnSetTensor4dDescriptor(FCTensorDesc,HIPDNN_TENSOR_NCHW,HIPDNN_DATA_FLOAT,
										in_n,1,FC_h,FC_w));
										
										
	float *pFC;
	hipMalloc(&pFC,in_h*in_w*FC_h*FC_w*sizeof(float));
	
	float *pFC_bias;
	hipMalloc(&pFC_bias,FC_h*FC_w*sizeof(float));


	float *FC_data;
	hipMalloc(&FC_data,in_n*FC_h*FC_w*sizeof(float));
	
				
	//Activation Functions
	hipdnnActivationDescriptor_t RELUActivation;
	hipdnnCreateActivationDescriptor(&RELUActivation);
	hipdnnSetActivationDescriptor(RELUActivation,HIPDNN_ACTIVATION_RELU,HIPDNN_PROPAGATE_NAN,0.0);
	hipdnnActivationDescriptor_t SIGActivation;
	hipdnnCreateActivationDescriptor(&SIGActivation);
	hipdnnSetActivationDescriptor(SIGActivation,HIPDNN_ACTIVATION_SIGMOID,HIPDNN_PROPAGATE_NAN,0.0);
	float *FC_relu_data;
	hipMalloc(&FC_relu_data,in_n*FC_h*FC_w*sizeof(float));
	
				
	//AE OUTPUT
	const int AE_out_h = in_h;
	const int AE_out_w = in_w;
	if(verbose == 1){
		std::cout << "AE_out_h: " << AE_out_h << std::endl;
		std::cout << "AE_out_w: " << AE_out_w << std::endl;
		std::cout << std::endl;
	}
	
	hipdnnTensorDescriptor_t AE_outTensorDesc;
	CUDNN_CALL(hipdnnCreateTensorDescriptor(&AE_outTensorDesc));
	CUDNN_CALL(hipdnnSetTensor4dDescriptor(AE_outTensorDesc,HIPDNN_TENSOR_NCHW,HIPDNN_DATA_FLOAT,
										in_n,1,AE_out_h,AE_out_w));
										
	float *pOUT;
	hipMalloc(&pOUT,AE_out_h*AE_out_w*FC_h*FC_w*sizeof(float));
	
	float *pOUT_bias;
	hipMalloc(&pOUT_bias,AE_out_h*AE_out_w*sizeof(float));

	float *AE_out_data;
	hipMalloc(&AE_out_data,in_n*AE_out_h*AE_out_w*sizeof(float));
	
	//onevec
	float *onevec;
	hipMalloc(&onevec,in_n*sizeof(float));
	
	
	//perform
	float alpha = 1.f;
	float beta = 0.f;
	//cast double to float
	doubleTofloat<<<in_h*in_w,in_n>>>(d_A,in_data);
	//FC weight data
	doubleTofloat<<<in_h * in_w * FC_h * FC_w,1>>>(d_weights_in[0],pFC);
	//FC_bias weight data
	doubleTofloat<<<FC_h * FC_w,1>>>(d_weights_in[1],pFC_bias);
	//Fill onevec
	dev_const<<<in_n,1>>>(onevec,1);
	//input data to FC layer
	hipblasSgemm(cublasHandle,HIPBLAS_OP_T,HIPBLAS_OP_N,FC_w*FC_h,in_n,in_w*in_h,&alpha,
				pFC,in_w*in_h,
				in_data,in_w*in_h,
				&beta,
				FC_data,FC_w*FC_h);
	//add bias to FC layer
	hipblasSgemm(cublasHandle,HIPBLAS_OP_N,HIPBLAS_OP_N,FC_h*FC_w,in_n,1,&alpha,
				pFC_bias,FC_h*FC_w,
				onevec,1,
				&alpha,
				FC_data,FC_w*FC_h);
				
	//FC ReLU layer activation
	hipdnnActivationForward(cudnn,RELUActivation,&alpha,FCTensorDesc,FC_data,&beta,FCTensorDesc,FC_relu_data);
	//Output weight data
	doubleTofloat<<<AE_out_h*AE_out_w * FC_h * FC_w,1>>>(d_weights_in[2],pOUT);
	//Output bias weight data
	doubleTofloat<<<AE_out_h*AE_out_w,1>>>(d_weights_in[3],pOUT_bias);
	//FC layer to output layer
	hipblasSgemm(cublasHandle,HIPBLAS_OP_T,HIPBLAS_OP_N,AE_out_w*AE_out_h,in_n,FC_w*FC_h,&alpha,
				pOUT,FC_h*FC_w,
				FC_relu_data,FC_w*FC_h,
				&beta,
				AE_out_data,AE_out_w*AE_out_h);
	//add bias to AE_OUT layer
	hipblasSgemm(cublasHandle,HIPBLAS_OP_N,HIPBLAS_OP_N,AE_out_h*AE_out_w,in_n,1,&alpha,
				pOUT_bias,AE_out_h*AE_out_w,
				onevec,1,
				&alpha,
				AE_out_data,AE_out_h*AE_out_w);
						
	
	
	//returning
	//FC weights
	//cast float to double
	floatTodouble<<<in_w*in_h,in_n>>>(AE_out_data,d_OUT);
	plhs[0] = mxGPUCreateMxArrayOnGPU(OUT);
	
	
	// finalizing
	CUDA_CALL(hipFree(in_data));
	CUDA_CALL(hipFree(pFC));
	CUDA_CALL(hipFree(pFC_bias));
	CUDA_CALL(hipFree(FC_data));
	CUDA_CALL(hipFree(FC_relu_data));
	CUDA_CALL(hipFree(pOUT));
	CUDA_CALL(hipFree(pOUT_bias));
	CUDA_CALL(hipFree(AE_out_data));
	CUDA_CALL(hipFree(onevec));
	CUDNN_CALL(hipdnnDestroyTensorDescriptor(AE_outTensorDesc));
	CUDNN_CALL(hipdnnDestroyTensorDescriptor(FCTensorDesc));
	CUDNN_CALL(hipdnnDestroyTensorDescriptor(in_desc));
	CUDNN_CALL(hipdnnDestroyActivationDescriptor(RELUActivation));
	CUDNN_CALL(hipdnnDestroyActivationDescriptor(SIGActivation));
	CUDNN_CALL(hipdnnDestroy(cudnn));
	hipblasDestroy(cublasHandle);
	mxGPUDestroyGPUArray(A);
	mxGPUDestroyGPUArray(weights_in[0]);
	mxGPUDestroyGPUArray(weights_in[1]);
	mxGPUDestroyGPUArray(weights_in[2]);
	mxGPUDestroyGPUArray(weights_in[3]);
	
}
